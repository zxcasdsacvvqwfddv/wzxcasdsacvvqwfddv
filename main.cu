#include "hip/hip_runtime.h"

#include "include/TurboFFT.h"
    
template <typename DataType, int if_ft, int if_err, int gpu_spec>
void test_turbofft( DataType* input_d, DataType* output_d, DataType* output_turbofft,
                    DataType* twiddle_d, DataType* checksum, std::vector<long long int> param, 
                    long long int bs, int thread_bs, int ntest, ProgramConfig &config){
    long long int N = (1 << param[0]), threadblock_bs, Ni, WorkerFFTSize;
    long long int logN = param[0];
    long long int shared_size[3], griddims[3], blockdims[3]; 
    DataType* inputs[3] = {input_d, output_d, output_d + N * bs};
    DataType* outputs[3] = {output_d, output_d + N * bs, output_d};
    int kernel_launch_times = param[1];
    float gflops, elapsed_time, mem_bandwidth;
    hipEvent_t fft_begin, fft_end;
    
    hipblasHandle_t handle;      
    TurboFFT_Kernel_Entry<DataType, if_ft, if_err, gpu_spec> entry;
    int M = 16;
    dim3 gridDim1((N + 255) / 256, bs / M, 1);
    for(int i = 0; i < kernel_launch_times; ++i){
        threadblock_bs = param[5 + i];
        Ni = (1 << param[2 + i]); 
        WorkerFFTSize = param[8 + i]; 
        shared_size[i] = Ni * threadblock_bs * sizeof(DataType);
        
        blockdims[i] = (Ni * threadblock_bs) / WorkerFFTSize;
        long long int shared_per_SM = config.smem_size * 1024;
        griddims[i] = min(config.sm_cnt * min((2048 / blockdims[i]), (shared_per_SM / shared_size[i])), 
                ((N * bs) + (Ni * threadblock_bs) - 1) / (Ni * threadblock_bs));
        
        griddims[i] = ((((N * bs) + (Ni * threadblock_bs) - 1) / (Ni * threadblock_bs))) / thread_bs;
    
        hipFuncAttributes attr;
        if(hipFuncSetAttribute(reinterpret_cast<const void*>(entry.turboFFTArr[logN][i]), hipFuncAttributeMaxDynamicSharedMemorySize, shared_size[i]))
        printf("Set DynamicSharedMem failed\n");
        if(hipFuncSetAttribute(reinterpret_cast<const void*>(entry.turboFFTArr[logN][i]), hipFuncAttributePreferredSharedMemoryCarveout, (shared_per_SM * 100) / (config.smem_capacity * 1024)))
        printf("Set smemCarveout failed\n");
        hipError_t get_attr_res = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(entry.turboFFTArr[logN][i] ));
        if(get_attr_res != 0)
        printf("get_attr_res = %d\n", get_attr_res);
    }
    
    hipEventCreate(&fft_begin);
    hipEventCreate(&fft_end);
    #pragma unroll
    for(int i = 0; i < kernel_launch_times; ++i){
        entry.turboFFTArr[logN][i]<<<griddims[i], blockdims[i], shared_size[i]>>>(inputs[i], outputs[i], twiddle_d, checksum, bs, thread_bs);
    }

    hipEventRecord(fft_begin);
    #pragma unroll
    for (int j = 0; j < ntest; ++j){
    
        #pragma unroll
        for(int i = 0; i < kernel_launch_times; ++i){
            entry.turboFFTArr[logN][i]<<<griddims[i], blockdims[i], shared_size[i]>>>(inputs[i], outputs[i], twiddle_d, checksum, bs, thread_bs);
            hipDeviceSynchronize();
        }
    
        hipDeviceSynchronize();
    }
    hipEventRecord(fft_end);
    hipEventSynchronize(fft_begin);
    hipEventSynchronize(fft_end);
    hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);
    elapsed_time = elapsed_time / ntest;
    gflops = 5 * N * log2f(N) * bs / elapsed_time * 1000 / 1000000000.f;
    mem_bandwidth = (float)(N * bs * sizeof(DataType) * 2) / (elapsed_time) * 1000.f / 1000000000.f;
    printf("turboFFT, %d, %d, %8.3f, %8.3f, %8.3f\n",  (int)log2f(N),  (int)log2f(bs), elapsed_time, gflops, mem_bandwidth);
    
    checkCudaErrors(hipMemcpy((void*)output_turbofft, (void*)outputs[kernel_launch_times - 1], N * bs * sizeof(DataType), hipMemcpyDeviceToHost));
}

template <typename DataType, int if_ft, int if_err, int gpu_spec>
void TurboFFT_main(ProgramConfig &config){
    DataType* input, *output_turbofft, *output_cufft;
    DataType* input_d, *output_d, *twiddle_d;
    int ntest = 10;

    std::vector<std::vector<long long int> > params;
    
    params = utils::load_parameters(config.param_file_path);

    DataType* checksum_d, *checksum_h;
    hipMalloc((void**)&checksum_d, sizeof(DataType) * 16384 * 2);
    checksum_h = (DataType*)calloc(16384 * 2, sizeof(DataType));
    DataType* dest = checksum_h;
    for(int i = 2; i <= (1 << 13); i *= 2){
        utils::getDFTMatrixChecksum(dest, i);
        dest += i;
    }
    hipMemcpy((void*)checksum_d, (void*)checksum_h, sizeof(DataType) * 16384 * 2, hipMemcpyHostToDevice);
    if(!config.if_bench){
        // Verification
        utils::initializeData<DataType>(input, input_d, output_d, output_turbofft, output_cufft, twiddle_d, config.N, config.bs_end);

        if(config.if_verify){
            test_turbofft<DataType, if_ft, if_err, gpu_spec>(input_d, output_d, output_turbofft, twiddle_d, checksum_d, params[config.logN], config.bs, config.thread_bs, 1, config);
            profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, config.N, config.bs, 1);            
            utils::compareData<DataType>(output_turbofft, output_cufft, config.N * config.bs, 1e-4);
        }
        // Profiling
        if(config.if_profile){
            long long int bs_begin = config.bs;
            for(int bs = bs_begin; bs <= config.bs_end; bs += config.bs_gap)
            profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, config.N, config.bs, ntest);
            
            for(int bs = bs_begin; bs <= config.bs_end; bs += config.bs_gap)
            test_turbofft<DataType, if_ft, if_err, gpu_spec>(input_d, output_d, output_turbofft, twiddle_d, checksum_d, params[config.logN], config.bs, config.thread_bs, ntest, config);
        }
    }
    
    if(config.if_bench){
        utils::initializeData<DataType>(input, input_d, output_d, output_turbofft, output_cufft, twiddle_d, 1 << 25, config.param_2 + 3);
        long long int N = 1;
        for(long long int logN = 1; logN <= 25; ++logN){
            N *= 2;
            long long int bs = 1;
            if(config.if_bench % 10 == 2) bs = bs << (config.param_1 - logN);
            for(int i = 0; i <= config.param_1 - logN; i += 1){
                if(config.if_bench > 10) profiler::cufft::test_cufft<DataType>(input_d, output_d, output_cufft, N, bs, ntest);
                else test_turbofft<DataType, if_ft, if_err, gpu_spec>(input_d, output_d, output_turbofft, twiddle_d, checksum_d, params[logN], bs, config.thread_bs, ntest, config);
                bs *= 2;
                if(config.if_bench % 10 == 2) break; 
            }
        }
    }
    hipFree(input_d);
    hipFree(output_d);
    hipFree(twiddle_d);
    free(input);
    free(output_cufft);
    free(output_turbofft);
}

int main(int argc, char *argv[]){
    ProgramConfig config;
    config.parseCommandLine(argc, argv);
    
    config.displayConfig();
    // Proceed with the rest of the program
    if(config.gpu == "T4"){
        if(config.datatype == 0) {
            if(config.if_ft == 0) TurboFFT_main<float2, 0, 0, 75>(config);
            else if(config.if_err == 0) TurboFFT_main<float2, 1, 0, 75>(config);
            else TurboFFT_main<float2, 1, 1, 75>(config);
        }
        else {
            if(config.if_ft == 0) TurboFFT_main<double2, 0, 0, 75>(config);
            else if(config.if_err == 0) TurboFFT_main<double2, 1, 0, 75>(config);
            else TurboFFT_main<double2, 1, 1, 75>(config);
        }
    } else {
        if(config.datatype == 0) {
            if(config.if_ft == 0) TurboFFT_main<float2, 0, 0, 80>(config);
            else if(config.if_err == 0) TurboFFT_main<float2, 1, 0, 80>(config);
            else TurboFFT_main<float2, 1, 1, 80>(config);
        }
        else {
            if(config.if_ft == 0) TurboFFT_main<double2, 0, 0, 80>(config);
            else if(config.if_err == 0) TurboFFT_main<double2, 1, 0, 80>(config);
            else TurboFFT_main<double2, 1, 1, 80>(config);
        }

    }
    
    return 0;
}


    